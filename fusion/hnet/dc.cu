#include "hip/hip_runtime.h"
/* 

    kernel for the dynamic chunking layer of HNet
*/

// each warp handles 16 bytes of the input
// blocks handle 16 * NUM_WARPS bytes of the input

// * denotes element wise multiplication

#include "kittens.cuh"
using namespace kittens;

// dimensions

constexpr int BATCH_DIM = 1;
constexpr int SEQ_LEN = 128;
constexpr int HEAD_DIM = 1024; // d_model = d_k 

// grid dimensions
#define NUM_WARPS 8
#define NUM_THREADS (NUM_WARPS * kittens::WARP_THREADS) // 8 * 32 threads per block
#define NUM_BLOCKS (SEQ_LEN / (NUM_WARPS * 16)) // 128 blocks @ 8 warps per block -> 1024 bytes per block

// inputs are bfloat16

struct dc_globals {
    using x_gl = gl<bf16, -1, -1, -1, -1, st_bf<16, 16>>; // 16x16 tile (loading 16 bytes per warp)
    using weights_gl = gl<bf16, -1, -1, -1, -1, st_bf<16, 16>>; // 16x16 tile corresponding to the input spatial location
    using p_gl = gl<bf16, -1, -1, -1, -1, sv_bf<16>>; // 16 element vector of p values corresponding to the 16 bytes of the block
    using b_gl = gl<bf16, -1, -1, -1, -1, sv_bf<16>>; // 16 element vector of b values

    x_gl x;
    weights_gl W_q, W_k;
    p_gl p;
    b_gl b;
}

__global__ __launch_bounds__(NUM_THREADS, 1)
void tk_dc(const __grid_constant__ dc_globals g) {

    // setup shared memory
    extern __shared__ alignment_dummy __shm[]; // allocates a dynamic amount of shared memory
    shared_allocator al((int*)&__shm[0]); // create a shared memory allocator and point it to the starting address of the shared memory

    int byte_id = blockIdx.x * NUM_WARPS + kittens::warpid(); // this is the chunk of 16 bytes that we're processing
    // block 0 warp 0 processing bytes 0-15
    // block 0 warp 1 processing bytes 16-31
    // block 1 warp 0 processing bytes 1024-1039

    // setup shared memory space for Q and K blocks
    st_bf<16, 16> (&W_q_s) = al.allocate<st_bf<16, 16>>(); // allocate memory for a 16x16 tile of W_q
    st_bf<16, 16> (&W_k_s) = al.allocate<st_bf<16, 16>>(); // allocate memory for a 16x16 tile of W_k

    // load the x and weights from HBM to shared
    load(g.x, g.x, {0, 0, byte_id, 0});

    // setup registers
    rt_bf<16, 16> x_r;
    rt_bf<16, 16, kittens::ducks::rt_layout::col> W_q_r;
    rt_bf<16, 16, kittens::ducks::rt_layout::col> W_k_r;

    rt_bf<16, 16> Q_r; // 16x16 tile of the Q block
    rt_bf<16, 16> K_r; // 16x16 tile of the K block

    rv_bf<16> cos_sim; // store the row wise dot products
    rv_bf<16> q_norm; // store the q row wise norm values
    rv_bf<16> k_norm; // store the k row wise norm values
    rv_bf<16> norm; // store the overall norm values
    rv_bf<16> p; // store the row wise p values

    // zero accumulators
    zero(Q_r);
    zero(K_r);
    zero(cos_sim);
    zero(q_norm);
    zero(k_norm);
    zero(norm);
    zero(p);

    // load x to registers
    load(x_r, g.x, {0, 0, byte_id, 0});

    // iterate over the HEAD_DIM dimension in chunks of 16
    for (int i = 0; i < HEAD_DIM; i += 16) {
        for (int j = 0; j < SEQ_LEN / 16; j++) {

            // load weights tiles from HBM to shared to register
            load(W_q_s, g.W_q, {0, 0, j, byte_id});
            load(W_q_r, W_q_s);

            load(W_k_s, g.W_k, {0, 0, j, byte_id});
            load(W_k_r, W_k_s);

            // multiply x with the weights and accumulate to Q_r and K_r
            mma_AB(Q_r, x_r, W_q_r, Q_r); // Q_r is our 16x16 accumulator which represents a tile of the Q block
            mma_AB(K_r, x_r, W_k_r, K_r); // K_r is our 16x16 accumulator which represents a tile of the K block
        }

        // similarity scores
        cos_sim = fma_AxBtC::op(Q_r, K_r, cos_sim); // Q_r * K_r + cos_sim 

        // norms
        q_norm = fma_AxBtC::op(Q_r, Q_r, q_norm); // Q_r * Q_r + q_norm
        k_norm = fma_AxBtC::op(K_r, K_r, k_norm); // K_r * K_r + k_norm

    }

    // convert to p_t scores
    // p_t = 1/2 (1 -(Q_t x K_(t-1)^T) / (||Q_t|| * ||K_(t-1)||) )
    // p = 1/2 (1 - cos_sim / (k_norm * q_norm))

    sqrt(q_norm, q_norm);
    sqrt(k_norm, k_norm);
    mul(norm, k_norm, q_norm); // k_norm * q_norm
    div(p, cos_sim, norm); // cos_sim / (norm)
    sub(p, 1, p); // 1 - p
    mul(p, p, 0.5); // 0.5 * (1 - p)

    store(p, g.p, {0, 0, byte_id, 0});

    rv_bf<16> b_r; // accumulator for boundary token values
    geq(b_r, p, bf16(0.5)); // b_r = p >= 0.5
    store(b_r, g.b, {0, 0, byte_id, 0}); // store the boundary token values

    // update the x values
    mul(x_r, x_r, b_r); // x_r = x_r * b_r, broadcasting is handled by TK
    store(x_r, g.x, {0, 0, byte_id, 0}); // store the updated x values

}

void dispatch_micro(float *d_x, float *d_W_q, float *d_W_k, float *d_p, float *d_b) {
    using globals = dc_globals;

    // create the global layouts
    globals::x_gl  x_arg{reinterpret_cast<__hip_bfloat16*>(d_x), 1, BATCH_DIM, SEQ_LEN, HEAD_DIM};
    globals::weights_gl  W_q_arg{reinterpret_cast<__hip_bfloat16*>(d_W_q), 1, BATCH_DIM, SEQ_LEN, HEAD_DIM};  
    globals::weights_gl  W_k_arg{reinterpret_cast<__hip_bfloat16*>(d_W_k), 1, BATCH_DIM, SEQ_LEN, HEAD_DIM};
    globals::p_gl  p_arg{reinterpret_cast<__hip_bfloat16*>(d_p), 1, 1, BATCH_DIM, SEQ_LEN};
    globals::b_gl  b_arg{reinterpret_cast<__hip_bfloat16*>(d_b), 1, 1, BATCH_DIM, SEQ_LEN};

    globals g{x_arg, W_q_arg, W_k_arg, p_arg, b_arg};

    unsigned long mem_size = 100960; 
    hipFuncSetAttribute(reinterpret_cast<const void*>(tk_dc), hipFuncAttributeMaxDynamicSharedMemorySize, mem_size);

    // Launch 1D grid with correct number of tiles
    tk_dc<<<NUM_BLOCKS, NUM_THREADS, mem_size>>>(g);
    hipDeviceSynchronize();
}
#include "harness.impl"
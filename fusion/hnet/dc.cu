#include "hip/hip_runtime.h"
/* 

    kernel for the dynamic chunking layer of HNet
*/

// each warp handles 16 bytes of the input
// blocks handle 16 * NUM_WARPS bytes of the input

// * denotes element wise multiplication
// accumulators in float to avoid overflows

#include "kittens.cuh"
#include "pyutils/pyutils.cuh"
using namespace kittens;

// dimensions
// commented out unused vars when running in Python

// constexpr int BATCH_DIM = 1;
constexpr int SEQ_LEN = 8192;
constexpr int HEAD_DIM = 1024; // d_model = d_k 

// grid dimensions
#define NUM_WARPS 8
#define NUM_THREADS (NUM_WARPS * kittens::WARP_THREADS) // 8 * 32 threads per block
#define NUM_BLOCKS (SEQ_LEN / (NUM_WARPS * 16)) // 128 blocks @ 8 warps per block -> 1024 bytes per block

// inputs are bfloat16

// define global layouts
using x_gl = gl<bf16, -1, -1, -1, -1, st_bf<16, 16>>; // 16x16 tile (loading 16 bytes per warp)
using weights_gl = gl<bf16, -1, -1, -1, -1, st_bf<16, 16>>; // 16x16 tile corresponding to the input spatial location
using p_gl = gl<bf16, -1, -1, -1, -1, sv_bf<16>>; // 16 element vector of p values corresponding to the 16 bytes of the block
using b_gl = gl<bf16, -1, -1, -1, -1, sv_bf<16>>; // 16 element vector of b values

struct dc_globals {
    // input vars
    x_gl x_q, x_k;
    weights_gl W_q, W_k;
    p_gl p;
    b_gl b;

    // grid - number of thread blocks we are launching
    dim3 grid() { return dim3(NUM_BLOCKS); }
    // block - number of threads in a thread block  
    dim3 block() { return dim3(NUM_THREADS); }
    // Safe shared memory size for H100
    size_t dynamic_shared_memory() { return 224000; }
};

// Sqrt operation following ThunderKittens pattern
struct sqrt_op {
    template<typename T> __device__ static inline T op(const T &a) { return sqrtf(a); }
};
template<> __device__ inline float2 sqrt_op::op<float2>(const float2 &a) { 
    return make_float2(sqrtf(a.x), sqrtf(a.y)); 
}
template<> __device__ inline bf16 sqrt_op::op<bf16>(const bf16 &a) { 
    return __float2bfloat16(sqrtf(__bfloat162float(a))); 
}

// B_t operation following ThunderKittens pattern  
struct b_t_op {
    template<typename T> __device__ static inline T op(const T &a) { 
        return a > 0.5f ? 1.0f : 0.0f; 
    }
};
template<> __device__ inline float2 b_t_op::op<float2>(const float2 &a) { 
    return make_float2(a.x > 0.5f ? 1.0f : 0.0f, a.y > 0.5f ? 1.0f : 0.0f); 
}
template<> __device__ inline bf16 b_t_op::op<bf16>(const bf16 &a) { 
    float f = __bfloat162float(a);
    return __float2bfloat16(f > 0.5f ? 1.0f : 0.0f); 
}

// Clamp operation: clamps values to [min_val, max_val]
struct clamp_op {
    template<typename T> __device__ static inline T op(const T &a) { 
        return max(min(a, 1.0f), 0.0f); 
    }
};
// Specialize for float2 (vectorized)
template<> __device__ inline float2 clamp_op::op<float2>(const float2 &a) { 
    return make_float2(
        max(min(a.x, 1.0f), 0.0f),
        max(min(a.y, 1.0f), 0.0f)
    ); 
}
// Specialize for bf16 (if needed, but we're clamping floats)
template<> __device__ inline bf16 clamp_op::op<bf16>(const bf16 &a) { 
    float f = __bfloat162float(a);
    f = max(min(f, 1.0f), 0.0f);
    return __float2bfloat16(f); 
}

__global__ __launch_bounds__(NUM_THREADS, 1)
void tk_dc(const __grid_constant__ dc_globals g) {

    // setup shared memory
    extern __shared__ alignment_dummy __shm[]; // allocates a dynamic amount of shared memory
    shared_allocator al((int*)&__shm[0]); // create a shared memory allocator and point it to the starting address of the shared memory

    int byte_id = blockIdx.x * NUM_WARPS + kittens::warpid(); // this is the chunk of 16 bytes that we're processing
    // block 0 warp 0 processing bytes 0-15
    // block 0 warp 1 processing bytes 16-31
    // block 1 warp 0 processing bytes 1024-1039

    // setup shared memory space for Q and K blocks
    st_bf<16, 16> (&W_q_s) = al.allocate<st_bf<16, 16>>(); // allocate memory for a 16x16 tile of W_q
    st_bf<16, 16> (&W_k_s) = al.allocate<st_bf<16, 16>>(); // allocate memory for a 16x16 tile of W_k
    st_bf<16, 16> (&x_q_s) = al.allocate<st_bf<16, 16>>(); // allocate memory for a 16x16 tile of x
    st_bf<16, 16> (&x_k_s) = al.allocate<st_bf<16, 16>>(); // allocate memory for a 16x16 tile of x
    __syncthreads();

    // setup registers
    rt_bf<16, 16> x_q_r; // 16x16 tile of the x values (seq x d_model chunk)
    rt_bf<16, 16> x_k_r; // 16x16 tile of the x values (seq x d_model chunk)
    rt_bf<16, 16, kittens::ducks::rt_layout::col> W_q_r; // 16x16 tile of the W_q values (d_model chunk x d_k chunk)
    rt_bf<16, 16, kittens::ducks::rt_layout::col> W_k_r; // 16x16 tile of the W_k values (d_model chunk x d_k chunk)

    rt_fl<16, 16> Q_r; // 16x16 accumulator for Q (seq x d_k chunk)
    rt_fl<16, 16> K_r; // 16x16 accumulator for K (seq x d_k chunk)

    using vec_t = typename decltype(Q_r)::col_vec; // use associated col_vec type for row reductions
    vec_t cos_sim; // store the row wise dot products (float for accumulation)
    vec_t q_norm; // store the q row wise norm values (float for accumulation)
    vec_t k_norm; // store the k row wise norm values (float for accumulation)
    vec_t norm; // store the overall norm values (float for computation)
    rv_bf<16> p; // store the row wise p values (can stay bf16 for final result)
    __syncthreads();

    // zero accumulators
    zero(Q_r);
    zero(K_r);
    zero(cos_sim);
    zero(q_norm);
    zero(k_norm);
    zero(norm);
    zero(p);

    // Tiled matrix multiplication for Q = x @ W_q and K = x @ W_k
    // Outer loop over d_k tiles (columns of output)
    for (int out_col = 0; out_col < HEAD_DIM / 16; out_col++) {
        // Reset partial accumulators for this output tile
        zero(Q_r);
        zero(K_r);
        __syncthreads();

        // Inner loop over d_model tiles (for contraction)
        for (int in_col = 0; in_col < HEAD_DIM / 16; in_col++) {
            // Load x tile: seq tile (byte_id) x d_model chunk (in_col)
            load(x_q_s, g.x_q, {0, 0, byte_id, in_col});
            load(x_q_r, x_q_s);
            load(x_k_s, g.x_k, {0, 0, byte_id, in_col});
            load(x_k_r, x_k_s);

            // Load W_q tile: d_model chunk (in_col) x d_k chunk (out_col) - note fixed seq=0 since weights are shared
            load(W_q_s, g.W_q, {0, 0, in_col, out_col});
            load(W_q_r, W_q_s);

            // Load W_k tile: same as above
            load(W_k_s, g.W_k, {0, 0, in_col, out_col});
            load(W_k_r, W_k_s);
            __syncthreads();

            // Accumulate into Q_r and K_r
            mma_AB(Q_r, x_q_r, W_q_r, Q_r);
            mma_AB(K_r, x_k_r, W_k_r, K_r);
            __syncthreads();
        }

        // Now compute partial contributions to cos_sim and norms for this output tile
        rt_fl<16, 16> el_wise_mul;
        __syncthreads();

        // Cosine sim partial: row_sum(Q_r * K_r)
        mul(el_wise_mul, Q_r, K_r);
        row_sum(cos_sim, el_wise_mul, cos_sim);
        __syncthreads();

        // Q norm partial: row_sum(Q_r * Q_r)
        mul(el_wise_mul, Q_r, Q_r);
        row_sum(q_norm, el_wise_mul, q_norm);
        __syncthreads();

        // K norm partial: row_sum(K_r * K_r)
        mul(el_wise_mul, K_r, K_r);
        row_sum(k_norm, el_wise_mul, k_norm);
        __syncthreads();
    }

    // convert to p_t scores
    // p_t = 1/2 (1 -(Q_t x K_(t-1)^T) / (||Q_t|| * ||K_(t-1)||) )
    // p = 1/2 (1 - cos_sim / (k_norm * q_norm))

    // add epsilon to norms to avoid division by zero
    add(q_norm, q_norm, 1e-12f);
    add(k_norm, k_norm, 1e-12f);
    __syncthreads();

    // Then use it with unary_op for vectors
    unary_op<sqrt_op>(q_norm, q_norm);  // sqrt(q_norm) -> q_norm
    unary_op<sqrt_op>(k_norm, k_norm);  // sqrt(k_norm) -> k_norm
    __syncthreads();

    mul(norm, k_norm, q_norm); // k_norm * q_norm
    __syncthreads();

    vec_t p_fl; // temporary vector for p calculation
    //add(norm, norm, 1e-12f); // add epsilon to avoid division by zero
    __syncthreads();

    div(p_fl, cos_sim, norm); // cos_sim / (norm)
    __syncthreads();


    sub(p_fl, p_fl, 1.0f); // p = p - 1
    __syncthreads();

    mul(p_fl, p_fl, -1.0f); // p = -(p - 1) = 1 - p
    __syncthreads();

    mul(p_fl, p_fl, 0.5f); // 0.5 * (1 - p)
    __syncthreads();

    // clamp p_fl to [0.0f, 1.0f]
    unary_op<clamp_op>(p_fl, p_fl); // this is what reduces our numerical error
    __syncthreads();

    copy(p, p_fl); // convert from float to bf16 for output
    __syncthreads();

    store(g.p, p, {0, 0, 0, byte_id});
    __syncthreads();

    rv_bf<16> b_r; // accumulator for boundary token values
    unary_op<b_t_op>(b_r, p); // b_r = p >= 0.5
    store(g.b, b_r, {0, 0, 0, byte_id}); // store the boundary token values

    // update the x values
    // we can't update the x values at this level b/c we've gone through the entire head dim but only have a 16x16 tile right now
    // best to do this on the client side

}

// Launch Kernel
void dispatch_dc(dc_globals g) {
    unsigned long mem_size = 50480; 
    hipFuncSetAttribute(reinterpret_cast<const void*>(
        tk_dc),
        hipFuncAttributeMaxDynamicSharedMemorySize,
        mem_size
    );
    tk_dc<<<g.grid(), g.block(), mem_size>>>(g);
    hipDeviceSynchronize();
}

/* OLD DISPATCH FUNCTION */

/* 
void dispatch_dc(float *d_x, float *d_W_q, float *d_W_k, float *d_p, float *d_b) {
    using globals = dc_globals;

    // create the global layouts
    globals::x_gl  x_arg{reinterpret_cast<__hip_bfloat16*>(d_x), 1, BATCH_DIM, SEQ_LEN, HEAD_DIM};
    globals::weights_gl  W_q_arg{reinterpret_cast<__hip_bfloat16*>(d_W_q), 1, BATCH_DIM, SEQ_LEN, HEAD_DIM};  
    globals::weights_gl  W_k_arg{reinterpret_cast<__hip_bfloat16*>(d_W_k), 1, BATCH_DIM, SEQ_LEN, HEAD_DIM};
    globals::p_gl  p_arg{reinterpret_cast<__hip_bfloat16*>(d_p), 1, 1, BATCH_DIM, SEQ_LEN};
    globals::b_gl  b_arg{reinterpret_cast<__hip_bfloat16*>(d_b), 1, 1, BATCH_DIM, SEQ_LEN};

    globals g{x_arg, W_q_arg, W_k_arg, p_arg, b_arg};

    unsigned long mem_size = 100960; 
    hipFuncSetAttribute(reinterpret_cast<const void*>(tk_dc), hipFuncAttributeMaxDynamicSharedMemorySize, mem_size);

    // Launch 1D grid with correct number of tiles
    tk_dc<<<NUM_BLOCKS, NUM_THREADS, mem_size>>>(g);
    hipDeviceSynchronize();
}

*/ 
PYBIND11_MODULE(tk_dc, m) {
    m.doc() = "tk_dc python module";
    // For wrapping kernels directly.
    kittens::py::bind_kernel<tk_dc, dc_globals>(m, "tk_dc", &dc_globals::x_q, &dc_globals::x_k, &dc_globals::W_q, &dc_globals::W_k, &dc_globals::p, &dc_globals::b);
    // For host functions that wrap the kernel, this will be called from Python
    kittens::py::bind_function<dispatch_dc, dc_globals>(m, "dispatch_dc", &dc_globals::x_q, &dc_globals::x_k, &dc_globals::W_q, &dc_globals::W_k, &dc_globals::p, &dc_globals::b);
}